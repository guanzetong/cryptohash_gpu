#include "hip/hip_runtime.h"
// Cryptohash

#include <wb.h>
#include <stdio.h>

#define NUM_CHAR 62
#define MAX_LEN 5

// CPU var
char *test;
uint8_t int_test[16];

// GPU constant memory
__constant__ uint32_t device_k[64];
__constant__ uint32_t device_r[64];
__constant__ uint32_t device_h_init[4];
__constant__ char device_charset[62];

// GPU functions

// leftrotate function definition
#define LEFTROTATE(x, c) (((x) << (c)) | ((x) >> (32 - (c))))

inline void to_bytes(uint32_t val, uint8_t *bytes)
{
    bytes[0] = (uint8_t) val;
    bytes[1] = (uint8_t) (val >> 8);
    bytes[2] = (uint8_t) (val >> 16);
    bytes[3] = (uint8_t) (val >> 24);
}

// HASH to uint8
void hash_to_int(char *charHash, uint8_t intHash[]){
    char tempChar[16][3];
    int j = 0;
    while(j < 16){
        tempChar[j][0] = charHash[j*2];
        tempChar[j][1] = charHash[j*2+1];
        tempChar[j][2] = '\0';
        ++j;
    }
    j = 0;
    while(j<16){
        sscanf(tempChar[j], "%x", (unsigned int*)(&(intHash[j])));
        ++j;
    }
}

// Convert 4 bytes(uint8_t) to 1 word(uint32_t)
__device__ uint32_t bytes_to_word(uint8_t *bytes)
{
    uint32_t word =  (uint32_t) bytes[0]
                  | ((uint32_t) bytes[1] << 8)
                  | ((uint32_t) bytes[2] << 16)
                  | ((uint32_t) bytes[3] << 24);
    return word;
}

// Convert 1 word(uint32_t) to 4 bytes(uint8_t)
__device__ void word_to_bytes(uint32_t word, uint8_t *bytes)
{
    bytes[0] = (uint8_t) word;
    bytes[1] = (uint8_t) (word >> 8);
    bytes[2] = (uint8_t) (word >> 16);
    bytes[3] = (uint8_t) (word >> 24);
}

// MD5
__device__ void md5(uint8_t *password, size_t length, uint8_t *digest) {

    // These vars will contain the hash
    uint32_t h0, h1, h2, h3;
    uint32_t w[16];
    uint32_t a, b, c, d, i, f, g, temp;

    // Append the "1" bit; most significant bit is "first"
    password[length] = 0x80;

    // Store password to register
    for (i = 0; i < 14; i++) {
        w[i] = bytes_to_word(password + i*4);
    }

    // Append the length in bits at the end of the buffer.
    uint8_t length_bytes[4];
    word_to_bytes(length<<3, length_bytes);
    w[14] = bytes_to_word(length_bytes); // the lower 4 bytes
    // length>>29 == length*8>>32, but avoids overflow.
    word_to_bytes(length>>29, length_bytes);
    w[15] = bytes_to_word(length_bytes); // the higher 4 bytes

    // Initialize variables - simple count in nibbles:
    h0 = device_h_init[0];
    h1 = device_h_init[1];
    h2 = device_h_init[2];
    h3 = device_h_init[3];

    // Initialize hash value for this chunk:
    a = h0;
    b = h1;
    c = h2;
    d = h3;

    // Main loop:
    for(i = 0; i<64; i++) {

        if (i < 16) {
            f = (b & c) | ((~b) & d);
            g = i;
        } else if (i < 32) {
            f = (d & b) | ((~d) & c);
            g = (5*i + 1) % 16;
        } else if (i < 48) {
            f = b ^ c ^ d;
            g = (3*i + 5) % 16;
        } else {
            f = c ^ (b | (~d));
            g = (7*i) % 16;
        }

        temp = d;
        d = c;
        c = b;
        b = b + LEFTROTATE((a + f + device_k[i] + w[g]), device_r[i]);
        a = temp;

    }

    // Add this chunk's hash to result so far:
    h0 += a;
    h1 += b;
    h2 += c;
    h3 += d;

    //var char digest[16] := h0 append h1 append h2 append h3 //(Output is in little-endian)
    word_to_bytes(h0, digest);
    word_to_bytes(h1, digest + 4);
    word_to_bytes(h2, digest + 8);
    word_to_bytes(h3, digest + 12);
}

// Password generator
__device__ void generate_password(size_t length, uint8_t *password, int id) {
    uint32_t current_word = id;
    for (int i = 0; i < length; i++) {
        password[length - 1 - i] = (uint8_t)device_charset[current_word % NUM_CHAR];
        current_word /= NUM_CHAR;
    }
    for (int i = length; i < 56; i++) {
        password[i] = 0;
    }
}

// Brute force
__global__ void brute_force(uint8_t *test_digest, size_t length, uint32_t max_num, uint8_t* found_flag, uint8_t *password_found) {

    uint8_t digest[16]; // This var will store the calculated MD5 hash
    uint8_t mismatch = 0; // The result of comparison between target hash and calculated hash
    uint8_t password[56]; // The password for calculation

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    for (int i = 0; tid + i < max_num; i = i + blockDim.x * gridDim.x) {

        // Generate a password for MD5
        generate_password(length, password, tid + i);

        // Invoke MD5
        md5(password, length, digest);

        // Compare with the target hash
        mismatch = 0;
        for (int j = 0; j < 16; j++) {
            if (digest[j] != test_digest[j]) {
                mismatch = 1;
            }
        }

        // If found the correct password, write to global memory
        if (mismatch == 0) {
            for (int j = 0; j < 56; j++) {
                password_found[j] = password[j];
            }
            *found_flag = 1;
        }

        __syncthreads();

        // Check if any thread has found the correct password
        // The found_flag is in global memory
        if (*found_flag == 1) {
            return;
        }
    }
}


__global__ void get_md5(uint8_t *password, size_t length, uint8_t *digest) {
    md5(password, length, digest);
}

__global__ void gen_pwd(int length, int max_num, uint8_t *password) {//, int *id, char *char_d) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    // uint8_t local_password[56];
    for (int i = 0; i + tid < max_num; i = i + blockDim.x * gridDim.x) {
        uint32_t password_index = (tid + i) * 56;
        uint32_t id_index = tid + i;
        generate_password(length, password + password_index, id_index);
    }
}

int main(int argc, char **argv) {

    // Check input parameters
    if (argc < 2) {
        printf("usage: %s 'stringhash'\n", argv[0]);
        return 1;
    }
    test = argv[1];
    if(strlen(test) != 32){
        printf ("Invalid hash. Exiting.\n");
        exit(0);
    }

    printf("Convert hash to uint8. Hash: ");
    hash_to_int(test, int_test);  // Convert target hash to uint8_t
    for (int i = 0; i < 32; i++) {
        printf("%c", test[i]);
    }
    printf("\n");

    printf("Declare variables\n");
    uint8_t *host_found_flag;
    uint8_t *host_password_found;

    uint8_t *device_found_flag;
    uint8_t *device_test_digest;
    uint8_t *device_password_found;

    printf("Allocate host memory\n");
    host_found_flag = (uint8_t *)malloc(sizeof(uint8_t));
    host_password_found = (uint8_t *)malloc(56 * sizeof(uint8_t));

    printf("Allocate device global memory\n");
    hipMalloc((void **)&device_test_digest, 16 * sizeof(uint8_t));
    hipMalloc((void **)&device_found_flag, sizeof(uint8_t));
    hipMalloc((void **)&device_password_found, 56 * sizeof(uint8_t));


    printf("Copy from host to device memory\n");
    *host_found_flag = 0;
    hipMemcpy(device_found_flag, host_found_flag, sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(device_test_digest, int_test, 16 * sizeof(uint8_t), hipMemcpyHostToDevice);

    printf("Prepare constants in device constant memory\n");

    uint32_t host_k[64] = {
        0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee ,
        0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501 ,
        0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be ,
        0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821 ,
        0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa ,
        0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8 ,
        0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed ,
        0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a ,
        0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c ,
        0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70 ,
        0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05 ,
        0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665 ,
        0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039 ,
        0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1 ,
        0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1 ,
        0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391 };

    uint32_t host_r[64] =  {
        7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22,
        5,  9, 14, 20, 5,  9, 14, 20, 5,  9, 14, 20, 5,  9, 14, 20,
        4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23,
        6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21};

    uint32_t host_h_init[4] = {0x67452301, 0xefcdab89, 0x98badcfe, 0x10325476};

    const char* host_charset = "0123456789abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ";

    hipMemcpyToSymbol(HIP_SYMBOL(device_k), host_k, 64 * sizeof(uint32_t));
    hipMemcpyToSymbol(HIP_SYMBOL(device_r), host_r, 64 * sizeof(uint32_t));
    hipMemcpyToSymbol(HIP_SYMBOL(device_h_init), host_h_init, 4 * sizeof(uint32_t));
    hipMemcpyToSymbol(HIP_SYMBOL(device_charset), host_charset, 62 * sizeof(char));


    printf("Scan from length = 1 to %d\n", MAX_LEN);
    printf("\n");
    printf("\n");
    int max_num = 1;
    for (int i = 0; i < MAX_LEN; i++) {
        max_num *= NUM_CHAR;
        size_t password_length = i + 1;
        printf("Password length: %d\n", i+1);
        printf("Number of combinations: %d\n", max_num);

        // Grid dimensions and block dimensions
        dim3 block_dim(256, 1, 1);
        dim3 grid_dim(256, 1, 1);
        
        // Invoke brute_force
        printf("Brute force start\n");
        brute_force<<<grid_dim, block_dim>>>(device_test_digest, password_length, max_num, device_found_flag, device_password_found);
        hipDeviceSynchronize();

        // Read result
        hipMemcpy(host_found_flag, device_found_flag, sizeof(uint8_t), hipMemcpyDeviceToHost);
        printf("Found flag: %d\n", *host_found_flag);
        printf("Brute force end\n");
        if (*host_found_flag == 1) {
            hipMemcpy(host_password_found, device_password_found, 56 * sizeof(uint8_t), hipMemcpyDeviceToHost);
            printf("The password is: ");
            for (int j = 0; j < i + 1; j++) {
                printf("%c", host_password_found[j]);
            }
            printf("\n");
            printf("\n");
            printf("\n");
            break;
        }

        printf("\n");
        printf("\n");

        // Scan through all given length
        if (i == MAX_LEN-1) {
            printf("The password is not found.\n");
        }
    }

    printf("Free host and device memory\n");
    hipFree(device_test_digest);
    hipFree(device_found_flag);
    hipFree(device_found_flag);
    free(host_found_flag);
    free(host_password_found);

    return 0;
}