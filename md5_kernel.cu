
#include <hip/hip_runtime.h>

// leftrotate function definition
#define LEFTROTATE(x, c) (((x) << (c)) | ((x) >> (32 - (c))))

typedef struct password_t{
    char word[56];
    size_t length;
} password;

// GPU constant memory
__constant__ uint32_t device_k[64];
__constant__ uint32_t device_r[64];
__constant__ uint32_t device_h_init[4];

// Convert 4 bytes(uint8_t) to 1 word(uint32_t)
__device__ uint32_t bytes_to_word(uint8_t *bytes)
{
    uint32_t word =  (uint32_t) bytes[0]
                  | ((uint32_t) bytes[1] << 8)
                  | ((uint32_t) bytes[2] << 16)
                  | ((uint32_t) bytes[3] << 24);
    return word;
}

// Convert 1 word(uint32_t) to 4 bytes(uint8_t)
__device__ void word_to_bytes(uint32_t word, uint8_t *bytes)
{
    bytes[0] = (uint8_t) word;
    bytes[1] = (uint8_t) (word >> 8);
    bytes[2] = (uint8_t) (word >> 16);
    bytes[3] = (uint8_t) (word >> 24);
}

// MD5
__device__ void md5(password *pwd, uint8_t *digest) {

    // These vars will contain the hash
    uint32_t h0, h1, h2, h3;
    uint32_t w[16];
    uint32_t a, b, c, d, i, f, g, temp;

    size_t init_len = pwd->length;
    uint8_t *msg = (uint8_t *)pwd->word;

    // Append the "1" bit; most significant bit is "first"
    msg[init_len] = 0x80;

    // Store password to register
    for (i = 0; i < 14; i++) {
        w[i] = bytes_to_word(msg + i*4);
    }

    // Append the length in bits at the end of the buffer.
    uint8_t length_bytes[4];
    word_to_bytes(init_len<<3, length_bytes);
    w[14] = bytes_to_word(length_bytes); // the lower 4 bytes
    // length>>29 == length*8>>32, but avoids overflow.
    word_to_bytes(init_len>>29, length_bytes);
    w[15] = bytes_to_word(length_bytes); // the higher 4 bytes

    // Initialize variables - simple count in nibbles:
    h0 = device_h_init[0];
    h1 = device_h_init[1];
    h2 = device_h_init[2];
    h3 = device_h_init[3];

    // Initialize hash value for this chunk:
    a = h0;
    b = h1;
    c = h2;
    d = h3;

    // Main loop:
    for(i = 0; i<64; i++) {

        if (i < 16) {
            f = (b & c) | ((~b) & d);
            g = i;
        } else if (i < 32) {
            f = (d & b) | ((~d) & c);
            g = (5*i + 1) % 16;
        } else if (i < 48) {
            f = b ^ c ^ d;
            g = (3*i + 5) % 16;
        } else {
            f = c ^ (b | (~d));
            g = (7*i) % 16;
        }

        temp = d;
        d = c;
        c = b;
        b = b + LEFTROTATE((a + f + device_k[i] + w[g]), device_r[i]);
        a = temp;

    }

    // Add this chunk's hash to result so far:
    h0 += a;
    h1 += b;
    h2 += c;
    h3 += d;

    //var char digest[16] := h0 append h1 append h2 append h3 //(Output is in little-endian)
    word_to_bytes(h0, digest);
    word_to_bytes(h1, digest + 4);
    word_to_bytes(h2, digest + 8);
    word_to_bytes(h3, digest + 12);
}
